#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include "Common.cuh"
#include "MatMulOnCPU.h"
using namespace std;

const int Row = 1024;
const int Col = 1024;
const int Blocksize = 32;

__global__ void myMatMulOnGPU(float* M, float* N, float* P, int width)
{
    int blockRow = blockIdx.x;
    int blockCol = blockIdx.y;
    int row = threadIdx.x;
    int col = threadIdx.y;

    float sum = 0;

    for (int i = 0; i < (width / Blocksize); i++) {
        __shared__ float Msub[Blocksize * Blocksize];
        __shared__ float Nsub[Blocksize * Blocksize];

        // each thread get 1 element from M and N
        Msub[col * Blocksize + row] = M[(i * Blocksize + col) * width + (blockRow * Blocksize + row)];
        Nsub[col * Blocksize + row] = N[(blockCol * Blocksize + col) * width + (i * Blocksize + row)];

        // make sure that the sub-matrices are loaded
        __syncthreads();

        // each thread compute 1 result
        for (int j = 0; j < Blocksize; j++)
            sum += Msub[j * Blocksize + row] * Nsub[col * Blocksize + j];
        
        // make sure that preceding computation is done
        // before the next iteration
        __syncthreads();
    }

    P[(blockCol * Blocksize + col) * width + (blockRow * Blocksize + row)] = sum;
}

int main()
{
    float* A = (float*)malloc(sizeof(float) * Row * Col);
    float* B = (float*)malloc(sizeof(float) * Row * Col);
    float* C = (float*)malloc(sizeof(float) * Row * Col);
    float* C_ref = (float*)malloc(sizeof(float) * Row * Col);
    mySetMatValue(A, Row, Col);
    mySetMatValue(B, Row, Col);
    myMatMulOnCPU(A, B, C_ref, Col);


    //malloc device memory
    float* d_dataA, * d_dataB, * d_dataC;
    myCudaDetermineGPU();
    CHECK(hipMalloc((void**)&d_dataA, sizeof(float) * Row * Col));
    CHECK(hipMalloc((void**)&d_dataB, sizeof(float) * Row * Col));
    CHECK(hipMalloc((void**)&d_dataC, sizeof(float) * Row * Col));
    CHECK(hipMemcpy(d_dataA, A, sizeof(float) * Row * Col, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_dataB, B, sizeof(float) * Row * Col, hipMemcpyHostToDevice));

    //init timing
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    //init block and grid dim
    dim3 threadPerBlock(Blocksize, Blocksize);
    dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y);
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);

    //warm up
    for (int i = 0; i < 10; i++)
        myMatMulOnGPU << <blockNumber, threadPerBlock >> > (d_dataA, d_dataB, d_dataC, Col);
    CHECK(hipGetLastError());

    CHECK(hipEventRecord(start, 0));
    myMatMulOnGPU << <blockNumber, threadPerBlock >> > (d_dataA, d_dataB, d_dataC, Col);
    CHECK(hipGetLastError());
    CHECK(hipEventRecord(stop, 0));

    //check result
    CHECK(hipMemcpy(C, d_dataC, sizeof(float) * Row * Col, hipMemcpyDeviceToHost));
    if (myMatCmp(C, C_ref, Row * Col))
    {
        printf("Error: Wrong result!\n");
        exit(-1);
    }

    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Total time: %fms\n", elapsedTime);

    //free resource
    free(A);
    free(B);
    free(C);
    CHECK(hipFree(d_dataA));
    CHECK(hipFree(d_dataB));
    CHECK(hipFree(d_dataC));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    return 0;
}