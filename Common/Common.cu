﻿#include "Common.cuh"
#include "hip/hip_runtime.h"
#include <iostream>

void myCudaDetermineGPU() {
    int numDevices;
    CHECK(hipGetDeviceCount(&numDevices));
    if (numDevices > 1) {
        int maxMultiprocessors = 0, maxDevice = 0;
        for (int device = 0; device < numDevices; device++) {
            hipDeviceProp_t props;
            CHECK(hipGetDeviceProperties(&props, device));
            if (maxMultiprocessors < props.multiProcessorCount) {
                maxMultiprocessors = props.multiProcessorCount;
                maxDevice = device;
            }
        }
        CHECK(hipSetDevice(maxDevice));
    }
}