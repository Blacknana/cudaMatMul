#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"
#include <stdio.h>
#include <iostream>
#include "Common.cuh"
#include "MatMulOnCPU.h"
using namespace std;

const int Row = 1024;
const int Col = 1024;

int main()
{
    float* A = (float*)malloc(sizeof(float) * Row * Col);
    float* B = (float*)malloc(sizeof(float) * Row * Col);
    float* C = (float*)malloc(sizeof(float) * Row * Col);
    float* C_ref = (float*)malloc(sizeof(float) * Row * Col);
    mySetMatValue(A, Row, Col);
    mySetMatValue(B, Row, Col);
    myMatMulOnCPU(A, B, C_ref, Col);


    myCudaDetermineGPU();
    hipblasHandle_t handle = 0;
    float alpha = 1, beta = 0;
    
    //malloc device memory
    float* d_dataA, * d_dataB, * d_dataC;
    CHECK_CUBLAS(hipblasCreate(&handle));
    CHECK(hipMalloc((void**)&d_dataA, sizeof(float) * Row * Col));
    CHECK(hipMalloc((void**)&d_dataB, sizeof(float) * Row * Col));
    CHECK(hipMalloc((void**)&d_dataC, sizeof(float) * Row * Col));
    CHECK_CUBLAS(hipblasSetMatrix(Row, Col, sizeof(float), A, Row, d_dataA, Row));
    CHECK_CUBLAS(hipblasSetMatrix(Row, Col, sizeof(float), B, Row, d_dataB, Row));

    //init timing
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    //warm up
    for (int i = 0; i < 10; i++)
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Row, Col, Col,
            &alpha, d_dataA, Row, d_dataB, Row, &beta, d_dataC, Row));

    CHECK(hipEventRecord(start, 0));
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Row, Col, Col,
        &alpha, d_dataA, Row, d_dataB, Row, &beta, d_dataC, Row));
    CHECK(hipEventRecord(stop, 0));

    //check result
    CHECK_CUBLAS(hipblasGetMatrix(Row, Col, sizeof(float), d_dataC, Row, C, Row));
    if (myMatCmp(C, C_ref, Row * Col))
    {
        printf("Error: Wrong result!\n");
        exit(-1);
    }

    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Total time: %fms\n", elapsedTime);

    //free resource
    free(A);
    free(B);
    free(C);
    CHECK(hipFree(d_dataA));
    CHECK(hipFree(d_dataB));
    CHECK(hipFree(d_dataC));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}